/**
 *
 * Master Thesis: GPU-based Multiscale Analysis of Volume Data
 *
 * Copyright (C) 2011 Dominik Meyer <meyerd@mytum.de>
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * as published by the Free Software Foundation; either version 2
 * of the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
 *
 */

#include "calculateLipschitz.h"

#include "lipschitz_kernel.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "timing.h"
#include "cutil_replacement.h"

using namespace std;
using namespace Volume;

namespace Cuda { namespace Lipschitz {

/**
  Calculates the lipschitz/h�lder exponent for each voxel.

  @param input The angles in the xz plane and the angle in y direction, the modulus maxima
 */
Volume::MultilevelVolumeContainer<float, 3>* calculateLipschitz(Volume::MultilevelVolumeContainer<float, 3>& input) {
	if(input.x.sSize != input.y.sSize || input.y.sSize != input.z.sSize)
		throw CudaLipschitzError("Input erroneous input sizes.");
	if(input.x.m_lLevels != input.y.m_lLevels || input.y.m_lLevels != input.z.m_lLevels)
		throw CudaLipschitzError("Input erroneous levels.");
	VolumeSize sSize = input.x.sSize;
	long lLevels = input.x.m_lLevels;
	MultilevelVolumeContainer<float, 3>* output = new MultilevelVolumeContainer<float, 3>(sSize, lLevels);

	// Allocate device memory for one level
	size_t devPtrStoragePitch = sizeof(hipPitchedPtr);
	hipPitchedPtr* devPtrModulusStorage = NULL;
	hipPitchedPtr* devPtrAnglesXZStorage = NULL;
	hipPitchedPtr* devPtrAnglesYStorage = NULL;
	cutilSafeCall(hipMallocPitch<hipPitchedPtr>(&devPtrModulusStorage, &devPtrStoragePitch, sizeof(hipPitchedPtr), lLevels));
	cutilSafeCall(hipMallocPitch<hipPitchedPtr>(&devPtrAnglesXZStorage, &devPtrStoragePitch, sizeof(hipPitchedPtr), lLevels));
	cutilSafeCall(hipMallocPitch<hipPitchedPtr>(&devPtrAnglesYStorage, &devPtrStoragePitch, sizeof(hipPitchedPtr), lLevels));
	//cutilSafeCall(hipMalloc<hipPitchedPtr>(&devPtrStorage, sizeof(hipPitchedPtr)*lLevels));
	hipPitchedPtr* devSrcModulusPtr = new hipPitchedPtr[lLevels];
	hipPitchedPtr* devSrcAnglesXZPtr = new hipPitchedPtr[lLevels];
	hipPitchedPtr* devSrcAnglesYPtr = new hipPitchedPtr[lLevels];
	hipPitchedPtr devDstPtr = {0};

	OUT_INFO("calculateLipschitz: Allocating device memory >=%li bytes ...\n", sizeof(float)*sSize.x*sSize.y*sSize.z*(lLevels+1)*3);
	for(long l = 0; l < lLevels; ++l) {
		cutilSafeCall(hipMalloc3D(&(devSrcModulusPtr[l]), make_hipExtent(sizeof(float)*sSize.x, sSize.y, sSize.z)));
		cutilSafeCall(hipMalloc3D(&(devSrcAnglesXZPtr[l]), make_hipExtent(sizeof(float)*sSize.x, sSize.y, sSize.z)));
		cutilSafeCall(hipMalloc3D(&(devSrcAnglesYPtr[l]), make_hipExtent(sizeof(float)*sSize.x, sSize.y, sSize.z)));
	}
	cutilSafeCall(hipMalloc3D(&devDstPtr, make_hipExtent(sizeof(float)*sSize.x, sSize.y, sSize.z)));
	cutilSafeCall(hipMemcpy2D(devPtrModulusStorage, devPtrStoragePitch, devSrcModulusPtr, sizeof(hipPitchedPtr), 
		sizeof(hipPitchedPtr), lLevels, hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy2D(devPtrAnglesXZStorage, devPtrStoragePitch, devSrcAnglesXZPtr, sizeof(hipPitchedPtr), 
		sizeof(hipPitchedPtr), lLevels, hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy2D(devPtrAnglesYStorage, devPtrStoragePitch, devSrcAnglesYPtr, sizeof(hipPitchedPtr), 
		sizeof(hipPitchedPtr), lLevels, hipMemcpyHostToDevice));
	//cutilSafeCall(hipMemcpy(devPtrStorage, devSrcModulusPtr, sizeof(hipPitchedPtr), hipMemcpyHostToDevice));

	dim3 threads_lipschitz(lipschitzBlockSize.x, lipschitzBlockSize.y);
	dim3 grid_lipschitz((sSize.x + lipschitzBlockSize.x - 1) / lipschitzBlockSize.x,
			(sSize.y * sSize.z + lipschitzBlockSize.y - 1) / lipschitzBlockSize.y);

	hipMemcpy3DParms devToHostCopyParams = {0};
	hipMemcpy3DParms hostToDevCopyParams = {0};

	// Copy data to device
	OUT_INFO("calculateLipschitz: Copying input data to device ...\n");
	for(long l = 0; l < lLevels; l++) {
		// Modulus
		hostToDevCopyParams.dstPtr = devSrcModulusPtr[l];
		hostToDevCopyParams.extent = make_hipExtent(sizeof(float)*sSize.x, sSize.y, sSize.z);
		hostToDevCopyParams.srcPtr = make_hipPitchedPtr(input.z.getLevel(l), sizeof(float)*sSize.x, sSize.y, sSize.z);
		hostToDevCopyParams.kind = hipMemcpyHostToDevice;
		cutilSafeCall(hipMemcpy3D(&hostToDevCopyParams));
		// Angles XZ
		hostToDevCopyParams.dstPtr = devSrcAnglesXZPtr[l];
		hostToDevCopyParams.extent = make_hipExtent(sizeof(float)*sSize.x, sSize.y, sSize.z);
		hostToDevCopyParams.srcPtr = make_hipPitchedPtr(input.x.getLevel(l), sizeof(float)*sSize.x, sSize.y, sSize.z);
		hostToDevCopyParams.kind = hipMemcpyHostToDevice;
		cutilSafeCall(hipMemcpy3D(&hostToDevCopyParams));
		// Angles Y
		hostToDevCopyParams.dstPtr = devSrcAnglesYPtr[l];
		hostToDevCopyParams.extent = make_hipExtent(sizeof(float)*sSize.x, sSize.y, sSize.z);
		hostToDevCopyParams.srcPtr = make_hipPitchedPtr(input.y.getLevel(l), sizeof(float)*sSize.x, sSize.y, sSize.z);
		hostToDevCopyParams.kind = hipMemcpyHostToDevice;
		cutilSafeCall(hipMemcpy3D(&hostToDevCopyParams));
	}

	CTimer ctAllLevelsTime;
	CTimer ctOneLevelTime;
	CTimer ctOneStepTime;

	ctAllLevelsTime.Reset();

	for(long l = 0; l < lLevels; l++) {
		ctOneLevelTime.Reset();
		ctOneStepTime.Reset();
		OUT_INFO("calculateLipschitz: Running lipschitz kernel ... ");
		calculateLipschitz<float, 5><<<grid_lipschitz, threads_lipschitz>>>(devPtrModulusStorage, devPtrAnglesXZStorage,
			devPtrAnglesYStorage, devPtrStoragePitch / sizeof(hipPitchedPtr), devDstPtr, sSize, l, lLevels);
		hipDeviceSynchronize();
		cutilCheckMsg("calculateLipschitz: Kernel execution failed");
		devToHostCopyParams.dstPtr = make_hipPitchedPtr(output->x.getLevel(l), sizeof(float)*sSize.x, sSize.y, sSize.z);
		devToHostCopyParams.extent = make_hipExtent(sizeof(float)*sSize.x, sSize.y, sSize.z);
		devToHostCopyParams.srcPtr = devDstPtr;
		devToHostCopyParams.kind = hipMemcpyDeviceToHost;
		OUT_INFO("copy to cpu ... ");
		cutilSafeCall(hipMemcpy3D(&devToHostCopyParams));
		OUT_INFO("done [%.5fs].\n", (float)ctOneStepTime.Query());

		OUT_INFO("calculateLipschitz: Level %li done [%.5fs].\n", l, (float)ctOneLevelTime.Query());
	}
	DEBUG_OUT("calculateLipschitz: Total time (incl. copy) %.5fs.\n", (float)ctAllLevelsTime.Query());

	cutilSafeCall(hipFree(devPtrModulusStorage));
	cutilSafeCall(hipFree(devPtrAnglesXZStorage));
	cutilSafeCall(hipFree(devPtrAnglesYStorage));
	for(long l = 0; l < lLevels; ++l) {
		cutilSafeCall(hipFree(devSrcModulusPtr[l].ptr));
		cutilSafeCall(hipFree(devSrcAnglesXZPtr[l].ptr));
		cutilSafeCall(hipFree(devSrcAnglesYPtr[l].ptr));
	}
	SAFE_DELETE_ARRAY(devSrcModulusPtr);
	SAFE_DELETE_ARRAY(devSrcAnglesXZPtr);
	SAFE_DELETE_ARRAY(devSrcAnglesYPtr);
	cutilSafeCall(hipFree(devDstPtr.ptr));
	output->x.m_bIsOk = true;
	return output;
}
};};